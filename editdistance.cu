
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdint.h>

#define WINDOW_SIZE 16384

#define NUM_THREADS 128

#define CHUNK_SIZE_A WINDOW_SIZE / NUM_THREADS

#define CHUNK_SIZE_B 64

#define WINDOW_SIZE_B 4096

#define MAX_CORRESPONDING_WINDOWS_B 8

#define MAX_CORRESPONDING_CHUNKS_B 4

#define MOD_MAX_CORRESPONDING_CHUNKS_B 3

#define MAX_LENGTH 400000000

#define CHARS_PER_INT 16

#define CHAR_BITS 4

#define MAX_NUM_CHARS 16


char input_initial_1[MAX_LENGTH], input_initial_2[MAX_LENGTH];

uint64_t newinput_1[MAX_LENGTH / 64 * MAX_NUM_CHARS], newinput_2[MAX_LENGTH / CHARS_PER_INT];

int final_computation_1[MAX_LENGTH /  WINDOW_SIZE_B], final_computation_2[MAX_LENGTH / WINDOW_SIZE_B];

__constant__ uint64_t MAX_INT, HIGH_BIT;
__constant__ int NUM_CHARS;


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  #define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif


/////////////////////////////////////

// string alignment for two blocks
__device__ void myers_alignment(int length, int la, uint64_t *string_b, uint64_t *vn, uint64_t *vp, uint64_t *hn, uint64_t *hp, uint64_t *d0, int &score, uint64_t *same) {

  for ( int i = 0 ; i < length ; i ++ ) {
    uint64_t ch = (string_b[(i >> 4)] >> ((i & 15) << 2)) & 15;


    uint64_t carry = 0, new_carry;
    uint64_t hp_shift = 1, hn_shift = 0;

    uint64_t *b = same;
    for ( int j = 0 ; j < la ; j ++ ) {
      uint64_t x = (b[ch] | vn[j]);
      uint64_t temp = x & vp[j];

      if ( temp > MAX_INT - carry || temp + carry > MAX_INT - vp[j] ) new_carry = 1; else new_carry = 0;
      temp += (carry + vp[j]);
      carry = new_carry;

      d0[j] = ((temp ^ vp[j])| x);
      hn[j] = vp[j] & d0[j];
      hp[j] = vn[j] | (~(vp[j] | d0[j]));

      uint64_t new_x = ((hp[j] << 1) | hp_shift) ;
      vn[j] = (new_x & d0[j]);
      vp[j] = (((hn[j] << 1) | hn_shift) | (~(new_x | d0[j])));

      hp_shift = ((hp[j] & HIGH_BIT) != 0);
      hn_shift = ((hn[j] & HIGH_BIT) != 0);

      b += NUM_CHARS;
    }

    if (hp_shift) score ++;
    else {
      if (hn_shift) score --;
    }

    
  }
}


__device__ int editdistance_gpu(char* str1, char* str2, int l1, int l2) {
    int* a = (int*)malloc((l2 + 1) * sizeof(int));
    int* b = (int*)malloc((l2 + 1) * sizeof(int));
    int* temp;

    for (int j = 0 ; j <= l2 ; j ++ ) a[j] = j;

    for (int i = 1; i <= l1; i++) {
        b[0] = i;
        for (int j = 1; j <= l2; j++) {
            b[j] = b[j - 1] + 1;
            if ( b[j] > a[j] + 1 ) b[j] = a[j] + 1;
            if (str1[i - 1] == str2[j - 1] && b[j] > a[j-1]) b[j] = a[j - 1];
            if (str1[i - 1] != str2[j - 1] && b[j] > a[j-1] + 1) b[j] = a[j - 1] + 1;

        }
        temp = a;
        a = b;
        b = temp;
    }
    return a[l2];
}



// compute edit distance for two strings
__global__ void new_edit_distance(int l2, uint64_t *input1,  uint64_t *input2,  uint64_t *solution, char *str_1, char *str_2) //, int work_length, int step_size, int shift, int NUM_POSSIBLE_END_POINTS)
{ 

   __shared__ int table[MAX_CORRESPONDING_WINDOWS_B][NUM_THREADS + 1][MAX_CORRESPONDING_CHUNKS_B];
   int temp_table[MAX_CORRESPONDING_WINDOWS_B];

   for ( int i = 0 ; i < MAX_CORRESPONDING_WINDOWS_B ; i ++ ) for ( int k = 0 ; k < MAX_CORRESPONDING_CHUNKS_B ; k ++ ) {
     table[i][threadIdx.x + 1][k] = 491510000;
     if ( threadIdx.x == 0 ) table[i][0][k] = 491510000;
   }
   table[0][0][MAX_CORRESPONDING_CHUNKS_B - 1] = 0;
   table[0][threadIdx.x + 1][ MAX_CORRESPONDING_CHUNKS_B - 1] = (threadIdx.x + 1) * CHUNK_SIZE_A;

   __syncthreads(); 

   int temp_shift_1 = (blockIdx.x * WINDOW_SIZE + threadIdx.x * CHUNK_SIZE_A) / 64;


   uint64_t vn[MAX_CORRESPONDING_CHUNKS_B][CHUNK_SIZE_A / 64], vp[MAX_CORRESPONDING_CHUNKS_B][CHUNK_SIZE_A / 64], hp[MAX_CORRESPONDING_CHUNKS_B][CHUNK_SIZE_A / 64], 
            hn[MAX_CORRESPONDING_CHUNKS_B][CHUNK_SIZE_A / 64], d0[MAX_CORRESPONDING_CHUNKS_B][CHUNK_SIZE_A / 64];
   int score[MAX_CORRESPONDING_CHUNKS_B];

   for ( int i = 0 ; i < MAX_CORRESPONDING_CHUNKS_B ; i ++ )  {
     for ( int j = 0 ; j < CHUNK_SIZE_A / 64 ; j ++ ) {
       vn[i][j] = 0;
       vp[i][j] = MAX_INT;
       hp[i][j] = 0;
       hn[i][j] = 0;
       d0[i][j] = 0;
     }
     score[i] =  491510000;
   }
   score[0] = CHUNK_SIZE_A;


   for (  int i = 0 ; i < l2 / CHUNK_SIZE_B; i ++ ) {

     for ( int j = 0 ; j < MAX_CORRESPONDING_CHUNKS_B ; j ++ ) {
       myers_alignment(CHUNK_SIZE_B, CHUNK_SIZE_A / 64, input2 + i * CHUNK_SIZE_B / CHARS_PER_INT, vn[j], vp[j], hn[j], hp[j], d0[j], score[j], input1 + temp_shift_1 * NUM_CHARS);
     }

     __syncthreads();



     for ( int j = 0 ; j < MAX_CORRESPONDING_WINDOWS_B ; j ++ ) { 
       temp_table[j] = table[j][threadIdx.x + 1][(i + MAX_CORRESPONDING_CHUNKS_B - 1) & MOD_MAX_CORRESPONDING_CHUNKS_B] + CHUNK_SIZE_B; // current chunk of B corresponds to empty in A
       for ( int k = 0 ; k < MAX_CORRESPONDING_CHUNKS_B ; k ++ ) {
         if ( temp_table[j] > table[j][threadIdx.x][k] + score[(k + 1) & MOD_MAX_CORRESPONDING_CHUNKS_B] ) 
           temp_table[j] = table[j][threadIdx.x][k] + score[(k + 1) & MOD_MAX_CORRESPONDING_CHUNKS_B];
       }
     }
     
     __syncthreads();
     


     for ( int j = 0 ; j < MAX_CORRESPONDING_WINDOWS_B ; j ++ ) {
       if ( threadIdx.x == 0 ) table[j][0][i & MOD_MAX_CORRESPONDING_CHUNKS_B] = table[j][0][(i + MAX_CORRESPONDING_CHUNKS_B - 1) & MOD_MAX_CORRESPONDING_CHUNKS_B] + CHUNK_SIZE_B;
       table[j][threadIdx.x + 1][i & MOD_MAX_CORRESPONDING_CHUNKS_B] = temp_table[j];
     }
     __syncthreads();

     for ( int j = 0 ; j < MAX_CORRESPONDING_WINDOWS_B ; j ++ ) {
       for ( int k = threadIdx.x ; k >= 0 && k >= threadIdx.x - 20 ; k -- ) { 
         if ( table[j][threadIdx.x + 1][i & MOD_MAX_CORRESPONDING_CHUNKS_B] > table[j][k][i & MOD_MAX_CORRESPONDING_CHUNKS_B] + (threadIdx.x + 1 - k) * CHUNK_SIZE_A ) 
           table[j][threadIdx.x + 1][i & MOD_MAX_CORRESPONDING_CHUNKS_B] = table[j][k][i & MOD_MAX_CORRESPONDING_CHUNKS_B] + (threadIdx.x + 1 - k) * CHUNK_SIZE_A;
       }
     }

     for ( int j = 0 ; j < CHUNK_SIZE_A / 64 ; j ++ ) {
       vn[(i + 1) % MAX_CORRESPONDING_CHUNKS_B][j] = 0;
       vp[(i + 1) % MAX_CORRESPONDING_CHUNKS_B][j] = MAX_INT;
       hp[(i + 1) % MAX_CORRESPONDING_CHUNKS_B][j] = 0;
       hn[(i + 1) % MAX_CORRESPONDING_CHUNKS_B][j] = 0;
       d0[(i + 1) % MAX_CORRESPONDING_CHUNKS_B][j] = 0;
     }
     score[(i + 1) % MAX_CORRESPONDING_CHUNKS_B] = CHUNK_SIZE_A;

     __syncthreads();

     if ( i % (WINDOW_SIZE_B / CHUNK_SIZE_B) == WINDOW_SIZE_B / CHUNK_SIZE_B - 1 ) { 
       if ( threadIdx.x == NUM_THREADS - 1 ) { 
         int v = i / (WINDOW_SIZE_B / CHUNK_SIZE_B);
         for ( int k = 0 ; k < MAX_CORRESPONDING_WINDOWS_B ; k ++ ) { 
           if ( v >= 0 ) {
             solution[blockIdx.x * (l2 / WINDOW_SIZE_B) * MAX_CORRESPONDING_WINDOWS_B + v * MAX_CORRESPONDING_WINDOWS_B + i / (WINDOW_SIZE_B / CHUNK_SIZE_B) - v] 
               = table[v % MAX_CORRESPONDING_WINDOWS_B][NUM_THREADS][i % MAX_CORRESPONDING_CHUNKS_B]; 
           }
           v--;
         }  
       }

       int v = (i / (WINDOW_SIZE_B / CHUNK_SIZE_B) + 1) % MAX_CORRESPONDING_WINDOWS_B;
       for ( int k = 0 ; k < MAX_CORRESPONDING_CHUNKS_B ; k ++ ) {
         table[v][threadIdx.x + 1][k] = 491510000;
         if ( threadIdx.x == 0 ) table[v][0][k] = 491510000;
       }
       table[v][0][MAX_CORRESPONDING_CHUNKS_B - 1] = 0;
       table[v][threadIdx.x + 1][ MAX_CORRESPONDING_CHUNKS_B - 1] = (threadIdx.x + 1) * CHUNK_SIZE_A;

       __syncthreads();
     }

  }
}



int editdistance_cpu(char* str1, char* str2, int l1, int l2) {
    int* a = (int*)malloc((l2 + 1) * sizeof(int));
    int* b = (int*)malloc((l2 + 1) * sizeof(int));
    int* temp;

    for (int j = 0 ; j <= l2 ; j ++ ) a[j] = j;

    for (int i = 1; i <= l1; i++) {
if ( i % 1000 == 0 ) printf("%d %d\n", i, l1);
        b[0] = i;
        for (int j = 1; j <= l2; j++) {
            b[j] = b[j - 1] + 1;
            if ( b[j] > a[j] + 1 ) b[j] = a[j] + 1;
            if (str1[i - 1] == str2[j - 1] && b[j] > a[j-1]) b[j] = a[j - 1];
            if (str1[i - 1] != str2[j - 1] && b[j] > a[j-1] + 1) b[j] = a[j - 1] + 1;

        }
        temp = a;
        a = b;
        b = temp;
    }
    return a[l2];
}


int main(void)
{
  // read file
   int cc = 0;
  freopen("file1.fasta", "r", stdin);
  char header1[500];
  fgets(header1, 500, stdin);
  while (scanf("%s", input_initial_1+cc) != EOF) {
     int t = strlen(input_initial_1+cc);
    cc += t;
  }

  freopen("file2.fasta", "r", stdin);
  char header2[500];
  fgets(header2, 500, stdin);
  cc = 0;
  while (scanf("%s", input_initial_2+cc) != EOF) {
     int t = strlen(input_initial_2+cc);
    cc += t;
  }

  int l1 = strlen(input_initial_1);
  int l2 = strlen(input_initial_2);

  printf("string 1 length = %d      string 2 length = %d\n", l1, l2);

  //count characters

   int used_chars[300];
   int char_count = 0;
  for (  int i = 0 ; i < 300; i ++ ) used_chars[i] = -1;
  for (  int i = 0 ; i < l1 ; i ++ ) 
    if ( used_chars[input_initial_1[i]] < 0 ) {
      char_count ++;
      used_chars[input_initial_1[i]] = char_count - 1;
    }
  
  for (  int i = 0 ; i < l2 ; i ++ ) 
    if ( used_chars[input_initial_2[i]] < 0 ) {
      char_count ++;
      used_chars[input_initial_2[i]] = char_count - 1;
    }
  if (char_count > (1 << CHAR_BITS) ) {
    printf("too many different characters!\n");
    return 0;
  }
  

  // convert input to int

  for ( int i = 0 ; i < l1 / 64 ; i ++ ) {
    for ( int j = 0 ; j < char_count ; j ++ ) {
      uint64_t s = 0;
      for ( int k = (i+1) * 64 - 1 ; k >= i * 64 ; k -- ) {
        s <<= 1;
        if ( used_chars[input_initial_1[k]] == j ) s += 1;
      }
      newinput_1[i * char_count + j] = s;
    }
  }
  for ( int i = 0 ; i < l2 / CHARS_PER_INT ; i ++ ) {
    newinput_2[i] = 0;
    for (  int j = 0 ; j < CHARS_PER_INT ; j ++ ) {
      newinput_2[i] |= (((uint64_t)used_chars[input_initial_2[i * CHARS_PER_INT + j]]) << (j * 4));
    }
  }


  //copy input to CUDA memory
  uint64_t *str1, *str2;

  hipMalloc(&str1, MAX_LENGTH / 64 * MAX_NUM_CHARS *sizeof(uint64_t)); 
  hipMalloc(&str2, MAX_LENGTH / CHARS_PER_INT *sizeof(uint64_t));
  hipMemcpy(str1, newinput_1, MAX_LENGTH / 64 * MAX_NUM_CHARS *sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(str2, newinput_2, MAX_LENGTH / CHARS_PER_INT*sizeof(uint64_t), hipMemcpyHostToDevice);




  char *real_str1, *real_str2;
  hipMalloc(&real_str1, MAX_LENGTH * sizeof(char));
  hipMalloc(&real_str2, MAX_LENGTH * sizeof(char));
  hipMemcpy(real_str1, input_initial_1, MAX_LENGTH * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(real_str2, input_initial_2, MAX_LENGTH * sizeof(char), hipMemcpyHostToDevice);





  const uint64_t max_int = UINT64_MAX;
  const uint64_t high_bit = ((uint64_t)1) << 63;
  hipMemcpyToSymbol(HIP_SYMBOL(MAX_INT), &max_int, sizeof(uint64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(HIGH_BIT), &high_bit, sizeof(uint64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(NUM_CHARS), &char_count, sizeof(int));

  int solution_size = 400000000;
  printf("GPU solution size = %d\n", solution_size);

  uint64_t *final_solution, *solution;
  hipMalloc(&solution, solution_size*sizeof(uint64_t));
  final_solution = (uint64_t*)malloc(solution_size*sizeof(uint64_t));
 
  printf("%d   %d    %d                  %d   %d   %d\n", WINDOW_SIZE, l1, l1 / WINDOW_SIZE, WINDOW_SIZE_B, l2, l2 / WINDOW_SIZE_B);

  printf("start parallel\n");


  
  new_edit_distance<<<l1 / WINDOW_SIZE, NUM_THREADS>>>(l2, str1, str2, solution, real_str1, real_str2); 
 


  hipError_t errSync  = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

  hipMemcpy(final_solution, solution, solution_size*sizeof( int), hipMemcpyDeviceToHost);

  printf("end parallel\n");

  // final solution computation

  int *s1, *s2, *temp_pointer; 
  s1 = final_computation_1;
  s2 = final_computation_2;
  s1[0] = 0;
  for ( int i = 1 ; i <= l2 / WINDOW_SIZE_B ; i ++ ) s1[i] = WINDOW_SIZE_B * i;

  for ( int i = 0 ; i < l1 / WINDOW_SIZE ; i ++ ) { 

     s2[0] = (i + 1) * WINDOW_SIZE;
     for ( int j = 0 ; j < l2 / WINDOW_SIZE_B ; j ++ ) {
       s2[j + 1] = s1[j + 1] + WINDOW_SIZE; // current window of A corresponds to empy in B

       for ( int k = 0 ; k < MAX_CORRESPONDING_WINDOWS_B ; k ++ ) {
         // current window of A corresponds to something non-trivial in B

         if ( j - k >= 0 && s2[j + 1] > s1[j - k] + final_solution[i * (l2 / WINDOW_SIZE_B) * MAX_CORRESPONDING_WINDOWS_B + (j - k) * MAX_CORRESPONDING_WINDOWS_B + k] )
           s2[j + 1] = s1[j - k] + final_solution[i * (l2 / WINDOW_SIZE_B) * MAX_CORRESPONDING_WINDOWS_B + (j - k) * MAX_CORRESPONDING_WINDOWS_B + k] ;


       }
     }
     temp_pointer = s1; s1 = s2; s2 = temp_pointer;     

  }
  printf("final edit distance = %d\n", s1[l2 / WINDOW_SIZE_B]);

  hipDeviceReset();
  hipFree(str1);
  hipFree(str2);
  hipFree(solution);
  free(final_solution);
}

